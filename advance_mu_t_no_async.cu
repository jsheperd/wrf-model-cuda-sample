#include "hip/hip_runtime.h"
// make sure 'jds==jps', etc.

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>
#include <string.h>
#include "config_flags.h"
#include "advance_mu_t_cu.h"

#define BLOCKSIZE 64
#define GPUs 3

#define min(a,b)  ((a)<(b)?(a):(b))

//int dev_id[GPUs] = {0};
//int dev_id[GPUs] = {0,2};
//int dev_id[GPUs] = {0,2,3};
int dev_id[GPUs] = {0,1,2};
//int dev_id[GPUs] = {0,1,2,3};

static void HandleError(hipError_t err,
                        const char *file,
                        int line) {
    if(err != hipSuccess){
      printf("%s in %s at line %d\n", hipGetErrorString(err),
             file, line);
        exit(EXIT_FAILURE);
    }
}

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

//--------------------------------------------------------------------
void advance_mu_t( float *ww, float *ww_1, float *u, float *u_1, 
                   float *v, float *v_1,            
                   float *mu, float *mut, float *muave, float *muts, 
                   float *muu,float *muv,
                   float *mudf, float *t, float *t_1,                      
                   float *t_ave, float *ft, float *mu_tend,                  
                   float rdx, float rdy, float dts, float epssm,               
                   float *dnw, float *fnm, float *fnp, float *rdnw,              
                   float *msfuy, float *msfvx_inv,                 
                   float *msftx, float *msfty,                  
                   config_flags config,                      
                   int ids, int ide, int jds, int jde, int kds, int kde,            
                   int ims, int ime, int jms, int jme, int kms, int kme,     
                   int its, int ite, int jts, int jte, int kts, int kte )
{
    size_t idim = ime-ims+1;
    size_t kdim = kme-kms+1;
    size_t jdim = jme-jms+1;

    dim3 Griddim(idim/BLOCKSIZE+1, jdim, 1);
    dim3 Blockdim(BLOCKSIZE, 1, 1);

    ide = ide - ids + 1;
    jde = jde - jds + 1;
    kde = kde - kds + 1; 
    ite = ite - its + 1; // i_size
    jte = jte - jts + 1; // j_size
    
    ids = ids - ims;
    jds = jds - jms;
    kds = kds - kms;
    its = its - ims; // i_start index in memory
    jts = jts - jms; // j_start index in memory
    
    ide = ide + ids - 1;
    jde = jde + jds - 1;
    kde = kde + kds - 1;
    ite = ite + its - 1; // i_end index in memory
    jte = jte + jts - 1; // j_end index in memory

    kte = kte - kts; 
    kts = 0;

    kme = kme - kms; 
    kms = 0;


    if((jts!=jds || jte != jde)){
      printf("jts!=jds || jte != jde\n");
      exit(1);
    }

// domain decomposition is performed equally in 'j' dimension 
// each GPU gets one 'j' row both before and after the output domain
    int start_address_2d[GPUs]; // 2d data's starting address on CPU for a GPU
    int start_address[GPUs]; // 3d data
    int start_address_2d_output[GPUs];
    int start_address_output[GPUs]; // output starting address doesn't need extra 'j' row
    int d_start_address_2d_output[GPUs];
    int d_start_address_output[GPUs];

    int num_rows[GPUs];
    int num_rows_output[GPUs];

    int jds_g[GPUs];
    int jde_g[GPUs];
    int jts_g[GPUs];
    int jte_g[GPUs];

    int total_rows = jme-jms+1;
    int acc_rows = 0; // accumulator for # of rows

    int j;
    if(GPUs == 1){
      num_rows[0] = total_rows;
      jds_g[0] = jds;
      jts_g[0] = jts; 
      jde_g[0] = jde;
      jte_g[0] = jte; 
      start_address[0] = 0;
      start_address_2d[0] = 0;
      start_address_output[0] = 0;
      start_address_2d_output[0] = 0;
      num_rows_output[0] = num_rows[0];
    }
    else
    for(j=0; j<GPUs; j++){
        if(j<GPUs-1){
          num_rows[j] = total_rows / GPUs;
        }
        else{
          num_rows[j] = total_rows - acc_rows;
        }

        if(j == 0){
          jds_g[j] = jds; // jts == jds -> set_physical_bc2d() is OK
          jts_g[j] = jts; 
          start_address[j] = 0;
          start_address_2d[j] = 0; 
        }
        else{
          jds_g[j] = 0; // if jds <=jts-1 then coriolis(), max( jds+1, jts ) is OK
          jts_g[j] = 3;
          start_address[j] = (acc_rows-3) * idim * kdim; // transfer three rows before 'jts'
          start_address_2d[j] = (acc_rows-3) * idim;
        }

        start_address_output[j] = (acc_rows) * idim * kdim;
        start_address_2d_output[j] = (acc_rows) * idim;
        acc_rows += num_rows[j];
        num_rows_output[j] = num_rows[j];

        if(j == 0){
          jde_g[j]=(num_rows[j]+3); // +3: three input row afters the first output row
          jte_g[j]=(num_rows[j]); // if jde >= jte+2 then min( jde-2, jte ) in coriolis() is OK
          num_rows[j] += 3;
        }
        else if(j == GPUs-1){
          jde_g[j]=(num_rows[j]+3); // +3: three input rows before the first output row
          jte_g[j]=(num_rows[j]+3); // jte == jde -> set_physical_bc2d() is OK
          num_rows[j] += 3;
        }
        else{
          jde_g[j]=(num_rows[j]+6); // +6: three input rows before the first output row, three input rows after the last output row
          jte_g[j]=(num_rows[j]+3) ; 
          num_rows[j] += 6; // three input rows before the first output row, three input rows after the last output row
        }
    }

#ifdef COALESCED
    size_t pitch_f;
#endif
    float *d_u[GPUs], *d_u_1[GPUs], *d_v[GPUs], *d_v_1[GPUs]; 
	float *d_t_1[GPUs], *d_ft[GPUs];
	float *d_ww[GPUs], *d_ww_1[GPUs], *d_t[GPUs], *d_t_ave[GPUs]; 
    float *d_mut[GPUs], *d_muu[GPUs], *d_muv[GPUs], *d_mu_tend[GPUs], *d_msfuy[GPUs]; 
    float *d_msfvx_inv[GPUs], *d_msftx[GPUs], *d_msfty[GPUs];
    float *d_mu[GPUs];
    float *d_muave[GPUs], *d_muts[GPUs], *d_mudf[GPUs];    
	float *d_dnw[GPUs], *d_fnm[GPUs], *d_fnp[GPUs], *d_rdnw[GPUs];
    float *d_wdtn[GPUs], *d_dvdxi[GPUs];
    float *d_dmdt[GPUs];	
	
    for(j=0; j<GPUs; j++){
        HANDLE_ERROR( hipSetDevice(dev_id[j]) );
        HANDLE_ERROR( hipDeviceSetCacheConfig(hipFuncCachePreferL1) );

#ifdef COALESCED
        HANDLE_ERROR( hipMallocPitch(&d_u[j]    , &pitch_f, idim * sizeof(float), kdim * num_rows[j]) );
        HANDLE_ERROR( hipMallocPitch(&d_u_1[j]  , &pitch_f, idim * sizeof(float), kdim * num_rows[j]) );
        HANDLE_ERROR( hipMallocPitch(&d_v[j]    , &pitch_f, idim * sizeof(float), kdim * num_rows[j]) );
        HANDLE_ERROR( hipMallocPitch(&d_v_1[j]  , &pitch_f, idim * sizeof(float), kdim * num_rows[j]) );
        HANDLE_ERROR( hipMallocPitch(&d_t_1[j]  , &pitch_f, idim * sizeof(float), kdim * num_rows[j]) );
        HANDLE_ERROR( hipMallocPitch(&d_ft[j]   , &pitch_f, idim * sizeof(float), kdim * num_rows[j]) );
        HANDLE_ERROR( hipMallocPitch(&d_ww[j]   , &pitch_f, idim * sizeof(float), kdim * num_rows[j]) );
        HANDLE_ERROR( hipMallocPitch(&d_ww_1[j] , &pitch_f, idim * sizeof(float), kdim * num_rows[j]) );
        HANDLE_ERROR( hipMallocPitch(&d_t[j]    , &pitch_f, idim * sizeof(float), kdim * num_rows[j]) );
        HANDLE_ERROR( hipMallocPitch(&d_t_ave[j], &pitch_f, idim * sizeof(float), kdim * num_rows[j]) );
        HANDLE_ERROR( hipMallocPitch(&d_dvdxi[j], &pitch_f, idim * sizeof(float), kdim * num_rows[j]) );
        HANDLE_ERROR( hipMallocPitch(&d_wdtn[j] , &pitch_f, idim * sizeof(float), kdim * num_rows[j]) );
       
        HANDLE_ERROR( hipMallocPitch(&d_mut[j]      , &pitch_f, idim * sizeof(float), num_rows[j]) );
        HANDLE_ERROR( hipMallocPitch(&d_muu[j]      , &pitch_f, idim * sizeof(float), num_rows[j]) );
        HANDLE_ERROR( hipMallocPitch(&d_muv[j]      , &pitch_f, idim * sizeof(float), num_rows[j]) );
        HANDLE_ERROR( hipMallocPitch(&d_mu_tend[j]  , &pitch_f, idim * sizeof(float), num_rows[j]) );
        HANDLE_ERROR( hipMallocPitch(&d_msfuy[j]    , &pitch_f, idim * sizeof(float), num_rows[j]) );
        HANDLE_ERROR( hipMallocPitch(&d_msfvx_inv[j], &pitch_f, idim * sizeof(float), num_rows[j]) );
        HANDLE_ERROR( hipMallocPitch(&d_msftx[j]    , &pitch_f, idim * sizeof(float), num_rows[j]) );
        HANDLE_ERROR( hipMallocPitch(&d_msfty[j]    , &pitch_f, idim * sizeof(float), num_rows[j]) );
        HANDLE_ERROR( hipMallocPitch(&d_mu[j]       , &pitch_f, idim * sizeof(float), num_rows[j]) );
        HANDLE_ERROR( hipMallocPitch(&d_muave[j]    , &pitch_f, idim * sizeof(float), num_rows[j]) );
        HANDLE_ERROR( hipMallocPitch(&d_muts[j]     , &pitch_f, idim * sizeof(float), num_rows[j]) );
        HANDLE_ERROR( hipMallocPitch(&d_mudf[j]     , &pitch_f, idim * sizeof(float), num_rows[j]) );
        HANDLE_ERROR( hipMallocPitch(&d_dmdt[j]     , &pitch_f, idim * sizeof(float), num_rows[j]) );

#else
        HANDLE_ERROR( hipMalloc(&d_u[j]    , idim * sizeof(float) * kdim * num_rows[j]) );
        HANDLE_ERROR( hipMalloc(&d_u_1[j]  , idim * sizeof(float) * kdim * num_rows[j]) );
        HANDLE_ERROR( hipMalloc(&d_v[j]    , idim * sizeof(float) * kdim * num_rows[j]) );
        HANDLE_ERROR( hipMalloc(&d_v_1[j]  , idim * sizeof(float) * kdim * num_rows[j]) );
        HANDLE_ERROR( hipMalloc(&d_t_1[j]  , idim * sizeof(float) * kdim * num_rows[j]) );
        HANDLE_ERROR( hipMalloc(&d_ft[j]   , idim * sizeof(float) * kdim * num_rows[j]) );
        HANDLE_ERROR( hipMalloc(&d_ww[j]   , idim * sizeof(float) * kdim * num_rows[j]) );
        HANDLE_ERROR( hipMalloc(&d_ww_1[j] , idim * sizeof(float) * kdim * num_rows[j]) );
        HANDLE_ERROR( hipMalloc(&d_t[j]    , idim * sizeof(float) * kdim * num_rows[j]) );
        HANDLE_ERROR( hipMalloc(&d_t_ave[j], idim * sizeof(float) * kdim * num_rows[j]) );
        HANDLE_ERROR( hipMalloc(&d_dvdxi[j], idim * sizeof(float) * kdim * num_rows[j]) );
        HANDLE_ERROR( hipMalloc(&d_wdtn[j] , idim * sizeof(float) * kdim * num_rows[j]) );
       
        HANDLE_ERROR( hipMalloc(&d_mut[j]      , idim * sizeof(float) * num_rows[j]) );
        HANDLE_ERROR( hipMalloc(&d_muu[j]      , idim * sizeof(float) * num_rows[j]) );
        HANDLE_ERROR( hipMalloc(&d_muv[j]      , idim * sizeof(float) * num_rows[j]) );
        HANDLE_ERROR( hipMalloc(&d_mu_tend[j]  , idim * sizeof(float) * num_rows[j]) );
        HANDLE_ERROR( hipMalloc(&d_msfuy[j]    , idim * sizeof(float) * num_rows[j]) );
        HANDLE_ERROR( hipMalloc(&d_msfvx_inv[j], idim * sizeof(float) * num_rows[j]) );
        HANDLE_ERROR( hipMalloc(&d_msftx[j]    , idim * sizeof(float) * num_rows[j]) );
        HANDLE_ERROR( hipMalloc(&d_msfty[j]    , idim * sizeof(float) * num_rows[j]) );
        HANDLE_ERROR( hipMalloc(&d_mu[j]       , idim * sizeof(float) * num_rows[j]) );
        HANDLE_ERROR( hipMalloc(&d_muave[j]    , idim * sizeof(float) * num_rows[j]) );
        HANDLE_ERROR( hipMalloc(&d_muts[j]     , idim * sizeof(float) * num_rows[j]) );
        HANDLE_ERROR( hipMalloc(&d_mudf[j]     , idim * sizeof(float) * num_rows[j]) );
        HANDLE_ERROR( hipMalloc(&d_dmdt[j]     , idim * sizeof(float) * num_rows[j]) );
        
#endif
        HANDLE_ERROR( hipMalloc(&d_dnw[j] , kdim * sizeof(float)) );
        HANDLE_ERROR( hipMalloc(&d_fnm[j] , kdim * sizeof(float)) );
        HANDLE_ERROR( hipMalloc(&d_fnp[j] , kdim * sizeof(float)) );
        HANDLE_ERROR( hipMalloc(&d_rdnw[j], kdim * sizeof(float)) );
         
    }
    for(j=0; j<GPUs; j++){
        HANDLE_ERROR( hipSetDevice(dev_id[j]) );
    // copy input from CPU to GPU
#ifdef COALESCED

        HANDLE_ERROR( hipMemcpy2D(d_u[j]    , pitch_f, &u[start_address[j]]    , idim * sizeof(float), idim * sizeof(float), kdim * num_rows[j], hipMemcpyHostToDevice) );
        HANDLE_ERROR( hipMemcpy2D(d_u_1[j]  , pitch_f, &u_1[start_address[j]]  , idim * sizeof(float), idim * sizeof(float), kdim * num_rows[j], hipMemcpyHostToDevice) );
        HANDLE_ERROR( hipMemcpy2D(d_v[j]    , pitch_f, &v[start_address[j]]    , idim * sizeof(float), idim * sizeof(float), kdim * num_rows[j], hipMemcpyHostToDevice) );
        HANDLE_ERROR( hipMemcpy2D(d_v_1[j]  , pitch_f, &v_1[start_address[j]]  , idim * sizeof(float), idim * sizeof(float), kdim * num_rows[j], hipMemcpyHostToDevice) );
        HANDLE_ERROR( hipMemcpy2D(d_t_1[j]  , pitch_f, &t_1[start_address[j]]  , idim * sizeof(float), idim * sizeof(float), kdim * num_rows[j], hipMemcpyHostToDevice) );
        HANDLE_ERROR( hipMemcpy2D(d_ft[j]   , pitch_f, &ft[start_address[j]]   , idim * sizeof(float), idim * sizeof(float), kdim * num_rows[j], hipMemcpyHostToDevice) );
        HANDLE_ERROR( hipMemcpy2D(d_ww[j]   , pitch_f, &ww[start_address[j]]   , idim * sizeof(float), idim * sizeof(float), kdim * num_rows[j], hipMemcpyHostToDevice) );
        HANDLE_ERROR( hipMemcpy2D(d_ww_1[j] , pitch_f, &ww_1[start_address[j]] , idim * sizeof(float), idim * sizeof(float), kdim * num_rows[j], hipMemcpyHostToDevice) );
        HANDLE_ERROR( hipMemcpy2D(d_t[j]    , pitch_f, &t[start_address[j]]    , idim * sizeof(float), idim * sizeof(float), kdim * num_rows[j], hipMemcpyHostToDevice) );
        HANDLE_ERROR( hipMemcpy2D(d_t_ave[j], pitch_f, &t_ave[start_address[j]], idim * sizeof(float), idim * sizeof(float), kdim * num_rows[j], hipMemcpyHostToDevice) );
       
        HANDLE_ERROR( hipMemcpy2D(d_mut[j]      , pitch_f, &mut[start_address_2d[j]]      , idim * sizeof(float), idim * sizeof(float), num_rows[j], hipMemcpyHostToDevice) );
        HANDLE_ERROR( hipMemcpy2D(d_muu[j]      , pitch_f, &muu[start_address_2d[j]]      , idim * sizeof(float), idim * sizeof(float), num_rows[j], hipMemcpyHostToDevice) );
        HANDLE_ERROR( hipMemcpy2D(d_muv[j]      , pitch_f, &muv[start_address_2d[j]]      , idim * sizeof(float), idim * sizeof(float), num_rows[j], hipMemcpyHostToDevice) );
        HANDLE_ERROR( hipMemcpy2D(d_mu_tend[j]  , pitch_f, &mu_tend[start_address_2d[j]]  , idim * sizeof(float), idim * sizeof(float), num_rows[j], hipMemcpyHostToDevice) );
        HANDLE_ERROR( hipMemcpy2D(d_msfuy[j]    , pitch_f, &msfuy[start_address_2d[j]]    , idim * sizeof(float), idim * sizeof(float), num_rows[j], hipMemcpyHostToDevice) );
        HANDLE_ERROR( hipMemcpy2D(d_msfvx_inv[j], pitch_f, &msfvx_inv[start_address_2d[j]], idim * sizeof(float), idim * sizeof(float), num_rows[j], hipMemcpyHostToDevice) );
        HANDLE_ERROR( hipMemcpy2D(d_msftx[j]    , pitch_f, &msftx[start_address_2d[j]]    , idim * sizeof(float), idim * sizeof(float), num_rows[j], hipMemcpyHostToDevice) );
        HANDLE_ERROR( hipMemcpy2D(d_msfty[j]    , pitch_f, &msfty[start_address_2d[j]]    , idim * sizeof(float), idim * sizeof(float), num_rows[j], hipMemcpyHostToDevice) );
        HANDLE_ERROR( hipMemcpy2D(d_mu[j]       , pitch_f, &mu[start_address_2d[j]]       , idim * sizeof(float), idim * sizeof(float), num_rows[j], hipMemcpyHostToDevice) );
        HANDLE_ERROR( hipMemcpy2D(d_muave[j]    , pitch_f, &muave[start_address_2d[j]]    , idim * sizeof(float), idim * sizeof(float), num_rows[j], hipMemcpyHostToDevice) );
        HANDLE_ERROR( hipMemcpy2D(d_muts[j]     , pitch_f, &muts[start_address_2d[j]]     , idim * sizeof(float), idim * sizeof(float), num_rows[j], hipMemcpyHostToDevice) );
        HANDLE_ERROR( hipMemcpy2D(d_mudf[j]     , pitch_f, &mudf[start_address_2d[j]]     , idim * sizeof(float), idim * sizeof(float), num_rows[j], hipMemcpyHostToDevice) );

#else

        HANDLE_ERROR( hipMemcpy(d_u[j]    , &u[start_address[j]]    , idim * sizeof(float) * kdim * num_rows[j], hipMemcpyHostToDevice) );
        HANDLE_ERROR( hipMemcpy(d_u_1[j]  , &u_1[start_address[j]]  , idim * sizeof(float) * kdim * num_rows[j], hipMemcpyHostToDevice) );
        HANDLE_ERROR( hipMemcpy(d_v[j]    , &v[start_address[j]]    , idim * sizeof(float) * kdim * num_rows[j], hipMemcpyHostToDevice) );
        HANDLE_ERROR( hipMemcpy(d_v_1[j]  , &v_1[start_address[j]]  , idim * sizeof(float) * kdim * num_rows[j], hipMemcpyHostToDevice) );
        HANDLE_ERROR( hipMemcpy(d_t_1[j]  , &t_1[start_address[j]]  , idim * sizeof(float) * kdim * num_rows[j], hipMemcpyHostToDevice) );
        HANDLE_ERROR( hipMemcpy(d_ft[j]   , &ft[start_address[j]]   , idim * sizeof(float) * kdim * num_rows[j], hipMemcpyHostToDevice) );
        HANDLE_ERROR( hipMemcpy(d_ww[j]   , &ww[start_address[j]]   , idim * sizeof(float) * kdim * num_rows[j], hipMemcpyHostToDevice) );
        HANDLE_ERROR( hipMemcpy(d_ww_1[j] , &ww_1[start_address[j]] , idim * sizeof(float) * kdim * num_rows[j], hipMemcpyHostToDevice) );
        HANDLE_ERROR( hipMemcpy(d_t[j]    , &t[start_address[j]]    , idim * sizeof(float) * kdim * num_rows[j], hipMemcpyHostToDevice) );
        HANDLE_ERROR( hipMemcpy(d_t_ave[j], &t_ave[start_address[j]], idim * sizeof(float) * kdim * num_rows[j], hipMemcpyHostToDevice) );
       
        HANDLE_ERROR( hipMemcpy(d_mut[j]      , &mut[start_address_2d[j]]      , idim * sizeof(float) * num_rows[j], hipMemcpyHostToDevice) );
        HANDLE_ERROR( hipMemcpy(d_muu[j]      , &muu[start_address_2d[j]]      , idim * sizeof(float) * num_rows[j], hipMemcpyHostToDevice) );
        HANDLE_ERROR( hipMemcpy(d_muv[j]      , &muv[start_address_2d[j]]      , idim * sizeof(float) * num_rows[j], hipMemcpyHostToDevice) );
        HANDLE_ERROR( hipMemcpy(d_mu_tend[j]  , &mu_tend[start_address_2d[j]]  , idim * sizeof(float) * num_rows[j], hipMemcpyHostToDevice) );
        HANDLE_ERROR( hipMemcpy(d_msfuy[j]    , &msfuy[start_address_2d[j]]    , idim * sizeof(float) * num_rows[j], hipMemcpyHostToDevice) );
        HANDLE_ERROR( hipMemcpy(d_msfvx_inv[j], &msfvx_inv[start_address_2d[j]], idim * sizeof(float) * num_rows[j], hipMemcpyHostToDevice) );
        HANDLE_ERROR( hipMemcpy(d_msftx[j]    , &msftx[start_address_2d[j]]    , idim * sizeof(float) * num_rows[j], hipMemcpyHostToDevice) );
        HANDLE_ERROR( hipMemcpy(d_msfty[j]    , &msfty[start_address_2d[j]]    , idim * sizeof(float) * num_rows[j], hipMemcpyHostToDevice) );
        HANDLE_ERROR( hipMemcpy(d_mu[j]       , &mu[start_address_2d[j]]       , idim * sizeof(float) * num_rows[j], hipMemcpyHostToDevice) );
        HANDLE_ERROR( hipMemcpy(d_muave[j]    , &muave[start_address_2d[j]]    , idim * sizeof(float) * num_rows[j], hipMemcpyHostToDevice) );
        HANDLE_ERROR( hipMemcpy(d_muts[j]     , &muts[start_address_2d[j]]     , idim * sizeof(float) * num_rows[j], hipMemcpyHostToDevice) );
        HANDLE_ERROR( hipMemcpy(d_mudf[j]     , &mudf[start_address_2d[j]]     , idim * sizeof(float) * num_rows[j], hipMemcpyHostToDevice) );

#endif
        HANDLE_ERROR( hipMemcpy(d_dnw[j] , dnw , kdim * sizeof(float), hipMemcpyHostToDevice) );
        HANDLE_ERROR( hipMemcpy(d_fnm[j] , fnm , kdim * sizeof(float), hipMemcpyHostToDevice) );
        HANDLE_ERROR( hipMemcpy(d_fnp[j] , fnp , kdim * sizeof(float), hipMemcpyHostToDevice) );
        HANDLE_ERROR( hipMemcpy(d_rdnw[j], rdnw, kdim * sizeof(float), hipMemcpyHostToDevice) );
    
    }

    for(j=0; j<GPUs; j++){
        if(j == 0){
          d_start_address_output[j] = 0;        
          d_start_address_2d_output[j] = 0;
        }
        else{
#ifdef COALESCED
          d_start_address_output[j] = 3 * pitch_f / sizeof(float) * kdim; // skip first three rows
          d_start_address_2d_output[j] = 3 * pitch_f / sizeof(float);
#else
          d_start_address_output[j] = 3 * idim * kdim;        
          d_start_address_2d_output[j] = 3 * idim;
#endif
        }
    }

    struct timeval ta, tb;
    long mseca, msecb;
    gettimeofday( &ta, NULL );
    mseca = ta.tv_sec * 1000000 + ta.tv_usec;
    
    for(j=0; j<GPUs; j++){
        HANDLE_ERROR( hipSetDevice(dev_id[j]) );
     
        advance_mu_t_kernel<<<Griddim, Blockdim>>>
                                    (d_ww[j], d_ww_1[j], d_u[j], d_u_1[j],
                                     d_v[j], d_v_1[j],
                                     d_mu[j], d_mut[j], d_muave[j], d_muts[j],
                                     d_muu[j], d_muv[j],
                                     d_mudf[j], d_t[j], d_t_1[j],
                                     d_t_ave[j], d_ft[j], d_mu_tend[j],
                                     rdx, rdy, dts, epssm,
                                     d_dnw[j], d_fnm[j], d_fnp[j], d_rdnw[j],
                                     d_msfuy[j], d_msfvx_inv[j],
                                     d_msftx[j], d_msfty[j],
                                     d_wdtn[j], d_dvdxi[j], d_dmdt[j],
                                     config,
                                     ids, ide, jds_g[j], jde_g[j], kds, kde,
#ifdef COALESCED
                                     pitch_f / sizeof(float), jdim, kdim,
#else
                                     idim, jdim, kdim,
#endif
                                     its, ite, jts_g[j], jte_g[j],
                                     kts,kte);
    }
    for(j=0; j<GPUs; j++){
        HANDLE_ERROR( hipSetDevice(dev_id[j]) );
        HANDLE_ERROR( hipDeviceSynchronize() );
    }


    gettimeofday( &tb, NULL );
    msecb = tb.tv_sec * 1000000 + tb.tv_usec;
    msecb -=mseca;
    printf("advance_mu_t GPU time is\t%.3f ms\n", (float)msecb/1000);

  // copy output from GPU to CPU
    for(j=0; j<GPUs; j++){
        HANDLE_ERROR( hipSetDevice(dev_id[j]) );

#ifdef COALESCED
        HANDLE_ERROR( hipMemcpy2D(&ww[start_address_output[j]]   , idim * sizeof(float), &d_ww[j][d_start_address_output[j]]   , pitch_f, idim * sizeof(float), kdim * num_rows_output[j], hipMemcpyDeviceToHost) );
        HANDLE_ERROR( hipMemcpy2D(&ww_1[start_address_output[j]] , idim * sizeof(float), &d_ww_1[j][d_start_address_output[j]] , pitch_f, idim * sizeof(float), kdim * num_rows_output[j], hipMemcpyDeviceToHost) );
        HANDLE_ERROR( hipMemcpy2D(&t[start_address_output[j]]    , idim * sizeof(float), &d_t[j][d_start_address_output[j]]      , pitch_f, idim * sizeof(float), kdim * num_rows_output[j], hipMemcpyDeviceToHost) );
        HANDLE_ERROR( hipMemcpy2D(&t_ave[start_address_output[j]], idim * sizeof(float), &d_t_ave[j][d_start_address_output[j]], pitch_f, idim * sizeof(float), kdim * num_rows_output[j], hipMemcpyDeviceToHost) );
        
        HANDLE_ERROR( hipMemcpy2D(&mu[start_address_2d_output[j]]   , idim * sizeof(float), &d_mu[j][d_start_address_2d_output[j]]   , pitch_f, idim * sizeof(float), num_rows_output[j], hipMemcpyDeviceToHost) );
        HANDLE_ERROR( hipMemcpy2D(&muave[start_address_2d_output[j]], idim * sizeof(float), &d_muave[j][d_start_address_2d_output[j]], pitch_f, idim * sizeof(float), num_rows_output[j], hipMemcpyDeviceToHost) );
        HANDLE_ERROR( hipMemcpy2D(&muts[start_address_2d_output[j]] , idim * sizeof(float), &d_muts[j][d_start_address_2d_output[j]] , pitch_f, idim * sizeof(float), num_rows_output[j], hipMemcpyDeviceToHost) );
        HANDLE_ERROR( hipMemcpy2D(&mudf[start_address_2d_output[j]] , idim * sizeof(float), &d_mudf[j][d_start_address_2d_output[j]] , pitch_f, idim * sizeof(float), num_rows_output[j], hipMemcpyDeviceToHost) );
#else
        HANDLE_ERROR( hipMemcpy(&ww[start_address_output[j]]   , &d_ww[j][d_start_address_output[j]]   , idim * sizeof(float) * kdim * num_rows_output[j], hipMemcpyDeviceToHost) );
        HANDLE_ERROR( hipMemcpy(&ww_1[start_address_output[j]] , &d_ww_1[j][d_start_address_output[j]] , idim * sizeof(float) * kdim * num_rows_output[j], hipMemcpyDeviceToHost) );
        HANDLE_ERROR( hipMemcpy(&t[start_address_output[j]]    , &d_t[j][d_start_address_output[j]]    , idim * sizeof(float) * kdim * num_rows_output[j], hipMemcpyDeviceToHost) );
        HANDLE_ERROR( hipMemcpy(&t_ave[start_address_output[j]], &d_t_ave[j][d_start_address_output[j]], idim * sizeof(float) * kdim * num_rows_output[j], hipMemcpyDeviceToHost) );
        
        HANDLE_ERROR( hipMemcpy(&mu[start_address_2d_output[j]]   , &d_mu[j][d_start_address_2d_output[j]]   , idim * sizeof(float) * num_rows_output[j], hipMemcpyDeviceToHost) );
        HANDLE_ERROR( hipMemcpy(&muave[start_address_2d_output[j]], &d_muave[j][d_start_address_2d_output[j]], idim * sizeof(float) * num_rows_output[j], hipMemcpyDeviceToHost) );
        HANDLE_ERROR( hipMemcpy(&muts[start_address_2d_output[j]] , &d_muts[j][d_start_address_2d_output[j]] , idim * sizeof(float) * num_rows_output[j], hipMemcpyDeviceToHost) );
        HANDLE_ERROR( hipMemcpy(&mudf[start_address_2d_output[j]] , &d_mudf[j][d_start_address_2d_output[j]] , idim * sizeof(float) * num_rows_output[j], hipMemcpyDeviceToHost) );
#endif
    }

    for(j=0; j<GPUs; j++){
        HANDLE_ERROR( hipSetDevice(dev_id[j]) );
        HANDLE_ERROR( hipFree( d_ww[j] ));
        HANDLE_ERROR( hipFree( d_ww_1[j] ));
        HANDLE_ERROR( hipFree( d_u[j] ));
        HANDLE_ERROR( hipFree( d_u_1[j] ));
        HANDLE_ERROR( hipFree( d_v[j] ));
        HANDLE_ERROR( hipFree( d_v_1[j] ));
        HANDLE_ERROR( hipFree( d_mu[j] ));
        HANDLE_ERROR( hipFree( d_mut[j] ));
        HANDLE_ERROR( hipFree( d_muave[j] ));
        HANDLE_ERROR( hipFree( d_muts[j] ));
        HANDLE_ERROR( hipFree( d_muu[j] ));
        HANDLE_ERROR( hipFree( d_muv[j] ));
        HANDLE_ERROR( hipFree( d_mudf[j] ));
        HANDLE_ERROR( hipFree( d_t[j] ));
        HANDLE_ERROR( hipFree( d_t_1[j] ));
        HANDLE_ERROR( hipFree( d_t_ave[j] ));
        HANDLE_ERROR( hipFree( d_ft[j] ));
        HANDLE_ERROR( hipFree( d_mu_tend[j] ));
        HANDLE_ERROR( hipFree( d_dnw[j] ));
        HANDLE_ERROR( hipFree( d_fnm[j] ));
        HANDLE_ERROR( hipFree( d_fnp[j] ));
        HANDLE_ERROR( hipFree( d_rdnw[j] ));
        HANDLE_ERROR( hipFree( d_msfuy[j] ));
        HANDLE_ERROR( hipFree( d_msfvx_inv[j] ));
        HANDLE_ERROR( hipFree( d_msftx[j] ));
        HANDLE_ERROR( hipFree( d_msfty[j] ));
        HANDLE_ERROR( hipFree( d_wdtn[j] ));
        HANDLE_ERROR( hipFree( d_dvdxi[j] ));
	    HANDLE_ERROR( hipFree( d_dmdt[j] ));
    }
}


